#include "hip/hip_runtime.h"
// libs
#include <nextweek/camera.cuh>
#include <nextweek/cbuffer.hpp>
#include <nextweek/color.hpp>
#include <nextweek/debug.hpp>
#include <nextweek/external.hpp>
#include <nextweek/hittables.cuh>
#include <nextweek/material.cuh>
#include <nextweek/ray.cuh>
#include <nextweek/sphere.cuh>
#include <nextweek/texture.cuh>
#include <nextweek/vec3.cuh>

__device__ Color ray_color(const Ray &r, Hittables **world,
                           hiprandState *local_rand_state,
                           int bounceNb) {
  Ray current_ray = r;
  Vec3 current_attenuation = Vec3(1.0f);
  while (bounceNb > 0) {
    HitRecord rec;
    bool anyHit =
        world[0]->hit(current_ray, 0.001f, FLT_MAX, rec);
    if (anyHit) {
      Ray scattered;
      Vec3 attenuation;
      bool isScattered = rec.mat_ptr->scatter(
          current_ray, rec, attenuation, scattered,
          local_rand_state);
      if (isScattered) {
        bounceNb--;
        current_attenuation *= attenuation;
        current_ray = scattered;
      } else {
        return Vec3(0.0f); // background color
      }
    } else {
      Vec3 udir = to_unit(current_ray.direction());
      float t = 0.5f * (udir.y() + 1.0f);
      Vec3 out = (1.0f - t) * Vec3(1.0f) +
                 t * Vec3(0.5f, 0.7f, 1.0f);
      return current_attenuation * out;
    }
  }
  return Vec3(0.0f); // background color
}

__global__ void rand_init(hiprandState *randState,
                          int seed) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprand_init(seed, 0, 0, randState);
  }
}

__global__ void render_init(int mx, int my,
                            hiprandState *randState,
                            int seed) {
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    hiprand_init(seed, 0, 0, randState);
  }
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if ((i >= mx) || (j >= my)) {
    return;
  }
  int pixel_index = j * mx + i;
  // same seed, different index
  hiprand_init(seed, pixel_index, 0,
              &randState[pixel_index]);
}

__global__ void render(Vec3 *fb, int maximum_x,
                       int maximum_y, int sample_nb,
                       int bounceNb, Camera dcam,
                       Hittables **world,
                       hiprandState *randState) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if ((i >= maximum_x) || (j >= maximum_y)) {
    return;
  }
  int pixel_index = j * maximum_x + i;
  hiprandState localS = randState[pixel_index];
  Vec3 rcolor(0.0f);
  Camera cam = dcam;
  for (int s = 0; s < sample_nb; s++) {
    float u = float(i + hiprand_uniform(&localS)) /
              float(maximum_x);
    float v = float(j + hiprand_uniform(&localS)) /
              float(maximum_y);
    Ray r = cam.get_ray(u, v, &localS);
    rcolor += ray_color(r, world, randState, bounceNb);
  }
  // fix the bounce depth
  randState[pixel_index] = localS;
  rcolor /= float(sample_nb);
  rcolor.e[0] = sqrt(rcolor.x());
  rcolor.e[1] = sqrt(rcolor.y());
  rcolor.e[2] = sqrt(rcolor.z());
  fb[pixel_index] = rcolor;
}

__global__ void make_world(Hittables **world, Hittable **ss,
                           int nx, int ny,
                           hiprandState *randState, int row,
                           unsigned char *imdata,
                           int *widths, int *heights,
                           int *bytes_per_pixels) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    // declare objects
    CheckerTexture *check =
        new CheckerTexture(Vec3(0.2, 0.8, 0.1));
    Lambertian *lamb = new Lambertian(check);
    ss[0] = new Sphere(Vec3(0, -1000.0, -1), 1000, lamb);
    int i = 1;
    int halfRow = row / 2;
    for (int a = -halfRow; a < halfRow; a++) {
      for (int b = -halfRow; b < halfRow; b++) {
        float choose_mat = hiprand_uniform(randState);
        Vec3 center(a + hiprand_uniform(randState), 0.2,
                    b + hiprand_uniform(randState));
        if (choose_mat < 0.8f) {
          Point3 center2 =
              center +
              Vec3(0, random_float(randState, 0.0, 0.5), 0);
          Color albedo = random_vec(randState);
          albedo *= random_vec(randState);
          Material *lamb1 = new Lambertian(albedo);
          ss[i++] = new MovingSphere(center, center2, 0.0,
                                     1.0, 0.2, lamb1);
        } else if (choose_mat < 0.95f) {

          Material *met = new Metal(
              Vec3(0.7f), 0.5f * hiprand_uniform(randState));
          ss[i++] = new Sphere(center, 0.2, met);
        } else {
          Material *diel = new Dielectric(1.5);
          ss[i++] = new Sphere(center, 0.2, diel);
        }
      }
    }

    Material *diel = new Dielectric(1.5);
    ss[i++] = new Sphere(Vec3(0, 1, 0), 1.0, diel);

    ImageTexture *imtex1 = new ImageTexture(
        imdata, widths, heights, bytes_per_pixels, 1);

    Material *lamb2 = new Lambertian(imtex1);
    ss[i++] = new Sphere(Vec3(-4, 1, 0), 1.3, lamb2);

    // ImageTexture *imtex2 = new ImageTexture(
    //    imdata, widths, heights, bytes_per_pixels, 0);
    NoiseTexture *ntxt = new NoiseTexture(4.3, randState);
    Material *met2 = new Lambertian(ntxt);
    // Material *met2 = new Metal(Vec3(0.1, 0.2, 0.5), 0.3);

    ss[i++] = new Sphere(Vec3(4, 1, 0), 1.0, met2);

    world[0] = new Hittables(ss, 22 * 22 + 1 + 3);
  }
}
__global__ void free_world(Hittables **world,
                           Hittable **ss) {
  int size = 22 * 22 + 1 + 3;
  for (int i = 0; i < size; i++) {
    delete ((Hittable *)ss[i])->mat_ptr;
    delete ss[i];
  }
  delete world[0];
}

int main() {
  float aspect_ratio = 16.0f / 9.0f;
  int WIDTH = 320;
  int HEIGHT = static_cast<int>(WIDTH / aspect_ratio);
  int BLOCK_WIDTH = 10;
  int BLOCK_HEIGHT = 10;
  int SAMPLE_NB = 30;
  int BOUNCE_NB = 20;

  std::cerr << "Resim boyutumuz " << WIDTH << "x" << HEIGHT
            << std::endl;

  std::cerr << BLOCK_WIDTH << "x" << BLOCK_HEIGHT
            << " bloklar halinde" << std::endl;

  // declare frame size
  int total_pixel_size = WIDTH * HEIGHT;
  size_t frameSize = 3 * total_pixel_size;

  // declare frame
  thrust::device_ptr<Vec3> fb =
      thrust::device_malloc<Vec3>(frameSize);
  CUDA_CONTROL(hipGetLastError());

  // declare random state
  int SEED = 1987;
  thrust::device_ptr<hiprandState> randState1 =
      thrust::device_malloc<hiprandState>(frameSize);
  CUDA_CONTROL(hipGetLastError());

  // declare random state 2
  thrust::device_ptr<hiprandState> randState2 =
      thrust::device_malloc<hiprandState>(1);
  CUDA_CONTROL(hipGetLastError());
  rand_init<<<1, 1>>>(thrust::raw_pointer_cast(randState2),
                      SEED);
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  // declare world
  thrust::device_ptr<Hittables *> world =
      thrust::device_malloc<Hittables *>(1);
  CUDA_CONTROL(hipGetLastError());
  int row = 22;
  int focus_obj_nb = 3;
  int nb_hittable = row * row + 1 + focus_obj_nb;
  thrust::device_ptr<Hittable *> hs =
      thrust::device_malloc<Hittable *>(nb_hittable);
  CUDA_CONTROL(hipGetLastError());

  // declara imdata
  std::vector<const char *> impaths = {"media/earthmap.png",
                                       "media/lsjimg.png"};
  std::vector<int> ws, hes, nbChannels;
  int totalSize;
  std::vector<unsigned char> imdata_h;
  imread(impaths, ws, hes, nbChannels, imdata_h, totalSize);
  // thrust::device_ptr<unsigned char> imda =
  //    thrust::device_malloc<unsigned char>(imd.size);
  unsigned char *h_ptr = imdata_h.data();

  // --------------------- image ------------------------
  unsigned char *imdata;
  CUDA_CONTROL(hipMalloc(&imdata, sizeof(unsigned char) *
                                       totalSize));
  CUDA_CONTROL(hipMemcpy((void *)imdata,
                          (const void *)h_ptr,
                          totalSize * sizeof(unsigned char),
                          hipMemcpyHostToDevice));

  std::size_t infosize = sizeof(int) * ws.size();

  int *imwidths;
  int *ws_ptr = ws.data();
  CUDA_CONTROL(hipMalloc(&imwidths, infosize));
  CUDA_CONTROL(hipMemcpy((void *)imwidths,
                          (const void *)ws_ptr, infosize,
                          hipMemcpyHostToDevice));

  int *imhs;
  int *hs_ptr = hes.data();
  CUDA_CONTROL(hipMalloc(&imhs, infosize));
  CUDA_CONTROL(hipMemcpy((void *)imhs,
                          (const void *)hs_ptr, infosize,
                          hipMemcpyHostToDevice));

  int *imch; // nb channels
  int *nb_ptr = nbChannels.data();
  CUDA_CONTROL(hipMalloc(&imch, infosize));
  CUDA_CONTROL(hipMemcpy((void *)imch,
                          (const void *)nb_ptr, infosize,
                          hipMemcpyHostToDevice));

  CUDA_CONTROL(hipGetLastError());

  make_world<<<1, 1>>>(thrust::raw_pointer_cast(world),
                       thrust::raw_pointer_cast(hs), WIDTH,
                       HEIGHT,
                       thrust::raw_pointer_cast(randState2),
                       row, imdata, imwidths, imhs, imch);
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  clock_t baslar, biter;
  baslar = clock();

  dim3 blocks(WIDTH / BLOCK_WIDTH + 1,
              HEIGHT / BLOCK_HEIGHT + 1);
  dim3 threads(BLOCK_WIDTH, BLOCK_HEIGHT);
  render_init<<<blocks, threads>>>(
      WIDTH, HEIGHT, thrust::raw_pointer_cast(randState1),
      SEED + 7);
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  // declare camera
  Vec3 lookfrom(13, 2, 3);
  Vec3 lookat(0, 0, 0);
  Vec3 wup(0, 1, 0);
  float vfov = 20.0f;
  float aspect_r = float(WIDTH) / float(HEIGHT);
  float dist_to_focus = 10.0;
  (lookfrom - lookat).length();
  float aperture = 0.1;
  float t0 = 0.0f, t1 = 1.0f;
  Camera cam(lookfrom, lookat, wup, vfov, aspect_r,
             aperture, dist_to_focus, t0, t1);

  render<<<blocks, threads>>>(
      thrust::raw_pointer_cast(fb), WIDTH, HEIGHT,
      SAMPLE_NB, BOUNCE_NB, cam,
      thrust::raw_pointer_cast(world),
      thrust::raw_pointer_cast(randState1));
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());
  biter = clock();
  double saniyeler =
      ((double)(biter - baslar)) / CLOCKS_PER_SEC;
  std::cerr << "Islem " << saniyeler << " saniye surdu"
            << std::endl;

  std::cout << "P3" << std::endl;
  std::cout << WIDTH << " " << HEIGHT << std::endl;
  std::cout << "255" << std::endl;

  for (int j = HEIGHT - 1; j >= 0; j--) {
    for (int i = 0; i < WIDTH; i++) {
      size_t pixel_index = j * WIDTH + i;
      thrust::device_reference<Vec3> pix_ref =
          fb[pixel_index];
      Vec3 pixel = pix_ref;
      int ir = int(255.99 * pixel.r());
      int ig = int(255.99 * pixel.g());
      int ib = int(255.99 * pixel.b());
      std::cout << ir << " " << ig << " " << ib
                << std::endl;
    }
  }
  CUDA_CONTROL(hipDeviceSynchronize());
  free_world<<<1, 1>>>(thrust::raw_pointer_cast(world),
                       thrust::raw_pointer_cast(hs));
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(fb);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(world);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(hs);
  CUDA_CONTROL(hipGetLastError());
  // dcam.free();
  hipFree(imdata);
  hipFree(imch);
  hipFree(imhs);
  hipFree(imwidths);
  // free(ws_ptr);
  // free(nb_ptr);
  // free(hs_ptr);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(randState2);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(randState1);
  CUDA_CONTROL(hipGetLastError());

  hipDeviceReset();
}
