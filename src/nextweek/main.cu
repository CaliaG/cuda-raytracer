#include "hip/hip_runtime.h"
// libs
#include <nextweek/camera.cuh>
#include <nextweek/cbuffer.hpp>
#include <nextweek/color.hpp>
#include <nextweek/debug.hpp>
#include <nextweek/external.hpp>
#include <nextweek/hittables.cuh>
#include <nextweek/kernels/makeworld.cuh>
#include <nextweek/kernels/trace.cuh>
#include <nextweek/material.cuh>
#include <nextweek/ray.cuh>
#include <nextweek/sphere.cuh>
#include <nextweek/texture.cuh>
#include <nextweek/vec3.cuh>

__global__ void rand_init(hiprandState *randState,
                          int seed) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprand_init(seed, 0, 0, randState);
  }
}

__global__ void render_init(int mx, int my,
                            hiprandState *randState,
                            int seed) {
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    hiprand_init(seed, 0, 0, randState);
  }
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if ((i >= mx) || (j >= my)) {
    return;
  }
  int pixel_index = j * mx + i;
  // same seed, different index
  hiprand_init(seed + pixel_index, pixel_index, 0,
              &randState[pixel_index]);
}

void get_device_props() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    std::cerr << "Device Number: " << i << std::endl;
    std::cerr << "Device name: " << prop.name << std::endl;
    std::cerr << "Memory Clock Rate (KHz): "
              << prop.memoryClockRate << std::endl;
    std::cerr << "Memory Bus Width (bits): "
              << prop.memoryBusWidth << std::endl;
    std::cerr << "  Peak Memory Bandwidth (GB/s): "
              << 2.0 * prop.memoryClockRate *
                     (prop.memoryBusWidth / 8) / 1.0e6
              << std::endl;
  }
}

int main() {
  float aspect_ratio = 16.0f / 9.0f;
  int WIDTH = 320;
  int HEIGHT = static_cast<int>(WIDTH / aspect_ratio);
  int BLOCK_WIDTH = 32;
  int BLOCK_HEIGHT = 18;
  int SAMPLE_NB = 100;
  int BOUNCE_NB = 50;

  get_device_props();

  std::cerr << "Resim boyutumuz " << WIDTH << "x" << HEIGHT
            << std::endl;

  std::cerr << BLOCK_WIDTH << "x" << BLOCK_HEIGHT
            << " bloklar halinde" << std::endl;

  // declare frame size
  int total_pixel_size = WIDTH * HEIGHT;
  size_t frameSize = 3 * total_pixel_size;

  // declare frame
  thrust::device_ptr<Vec3> fb =
      thrust::device_malloc<Vec3>(frameSize);
  CUDA_CONTROL(hipGetLastError());

  // declare random state
  int SEED = time(NULL);
  thrust::device_ptr<hiprandState> randState1 =
      thrust::device_malloc<hiprandState>(frameSize);
  CUDA_CONTROL(hipGetLastError());

  // declare random state 2
  thrust::device_ptr<hiprandState> randState2 =
      thrust::device_malloc<hiprandState>(1);
  CUDA_CONTROL(hipGetLastError());
  rand_init<<<1, 1>>>(thrust::raw_pointer_cast(randState2),
                      SEED);
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  // declare world
  thrust::device_ptr<Hittables *> world =
      thrust::device_malloc<Hittables *>(1);
  CUDA_CONTROL(hipGetLastError());
  int box_size = 6;
  int side_box_nb = 20;
  int sphere_nb = 10;
  int nb_hittable = side_box_nb;
  // nb_hittable *= side_box_nb;
  nb_hittable *= box_size;
  // nb_hittable += 1;
  thrust::device_ptr<Hittable *> hs =
      thrust::device_malloc<Hittable *>(nb_hittable);
  CUDA_CONTROL(hipGetLastError());

  // declara imdata
  // std::vector<const char *> impaths =
  // {"media/earthmap.png",
  //                                     "media/lsjimg.png"};
  // std::vector<int> ws, hes, nbChannels;
  // int totalSize;
  // std::vector<unsigned char> imdata_h;
  // imread(impaths, ws, hes, nbChannels, imdata_h,
  // totalSize);
  ////// thrust::device_ptr<unsigned char> imda =
  //////    thrust::device_malloc<unsigned char>(imd.size);
  // unsigned char *h_ptr = imdata_h.data();

  ////// --------------------- image
  ///------------------------
  // thrust::device_ptr<unsigned char> imdata;
  // upload_to_device(imdata, h_ptr, imdata_h.size());

  // int *ws_ptr = ws.data();

  // thrust::device_ptr<int> imwidths;
  // upload_to_device(imwidths, ws_ptr, ws.size());

  // thrust::device_ptr<int> imhs;
  // int *hs_ptr = hes.data();
  // upload_to_device(imhs, hs_ptr, hes.size());

  // thrust::device_ptr<int> imch; // nb channels
  // int *nb_ptr = nbChannels.data();
  // upload_to_device(imch, nb_ptr, nbChannels.size());

  // CUDA_CONTROL(hipGetLastError());

  make_world<<<1, 1>>>(
      thrust::raw_pointer_cast(world),
      thrust::raw_pointer_cast(hs),
      thrust::raw_pointer_cast(randState2), side_box_nb
      // thrust::raw_pointer_cast(imdata),
      // thrust::raw_pointer_cast(imwidths),
      // thrust::raw_pointer_cast(imhs),
      // thrust::raw_pointer_cast(imch)
      );
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  clock_t baslar, biter;
  baslar = clock();

  dim3 blocks(WIDTH / BLOCK_WIDTH + 1,
              HEIGHT / BLOCK_HEIGHT + 1);
  dim3 threads(BLOCK_WIDTH, BLOCK_HEIGHT);
  render_init<<<blocks, threads>>>(
      WIDTH, HEIGHT, thrust::raw_pointer_cast(randState1),
      SEED + 7);
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  // declare camera

  // one weekend final camera specification
  // Vec3 lookfrom(13, 2, 3);
  // Vec3 lookat(0, 0, 0);
  // Vec3 wup(0, 1, 0);
  // float vfov = 20.0f;
  // float aspect_r = float(WIDTH) / float(HEIGHT);
  // float dist_to_focus = 10.0;
  //(lookfrom - lookat).length();
  // float aperture = 0.1;
  // float t0 = 0.0f, t1 = 1.0f;

  // nextweek empty cornell box specification

  Vec3 lookfrom(478, 278, -600);
  Vec3 lookat(278, 278, 0);
  Vec3 wup(0, 1, 0);
  float vfov = 40.0f;
  float aspect_r = float(WIDTH) / float(HEIGHT);
  float dist_to_focus = (lookfrom - lookat).length();
  float aperture = 0.0;
  float t0 = 0.0f, t1 = 1.0f;

  Camera cam(lookfrom, lookat, wup, vfov, aspect_r,
             aperture, dist_to_focus, t0, t1);

  render<<<blocks, threads>>>(
      thrust::raw_pointer_cast(fb), WIDTH, HEIGHT,
      SAMPLE_NB, BOUNCE_NB, cam,
      thrust::raw_pointer_cast(world),
      thrust::raw_pointer_cast(randState1));
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());
  biter = clock();
  double saniyeler =
      ((double)(biter - baslar)) / CLOCKS_PER_SEC;
  std::cerr << "Islem " << saniyeler << " saniye surdu"
            << std::endl;

  std::cout << "P3" << std::endl;
  std::cout << WIDTH << " " << HEIGHT << std::endl;
  std::cout << "255" << std::endl;

  for (int j = HEIGHT - 1; j >= 0; j--) {
    for (int i = 0; i < WIDTH; i++) {
      size_t pixel_index = j * WIDTH + i;
      thrust::device_reference<Vec3> pix_ref =
          fb[pixel_index];
      Vec3 pixel = pix_ref;
      int ir = int(255.99 * pixel.r());
      int ig = int(255.99 * pixel.g());
      int ib = int(255.99 * pixel.b());
      std::cout << ir << " " << ig << " " << ib
                << std::endl;
    }
  }
  CUDA_CONTROL(hipDeviceSynchronize());
  CUDA_CONTROL(hipGetLastError());
  free_world(fb,    //
             world, //
             hs,    //
             // imdata, imch, imhs, imwidths, //
             randState1, //
             randState2);
  // free_world(fb, world, hs, randState1, randState2);
  CUDA_CONTROL(hipGetLastError());

  hipDeviceReset();
}
