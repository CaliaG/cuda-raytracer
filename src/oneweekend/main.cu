#include "hip/hip_runtime.h"
#include <oneweekend/vec3.cuh>
#include <oneweekend/color.hpp>
#include <oneweekend/ray.cuh>
#include <oneweekend/sphere.cuh>
#include <oneweekend/hittables.cuh>
#include <oneweekend/debug.hpp>
#include <oneweekend/external.hpp>


__device__ Color ray_color(const Ray & r, Hittables** world){
    HitRecord rec;
    bool anyHit = world[0]->hit(r, 0.0f, FLT_MAX, rec);
    if (anyHit){
        return 0.5f * Vec3(rec.normal.x() + 1.0f, 
                rec.normal.y()+1.0f,
                rec.normal.z() + 1.0f);
    } else {
        Vec3 udir = to_unit(r.direction());
        float t = 0.5f * (udir.y() + 1.0f);
        return (1.0f-t)*Vec3(1.0f)+ t*Vec3(0.5f, 0.7f, 1.0f);
    }
}

__global__ void render(Vec3 *fb, int maximum_x, int maximum_y,
        Vec3 lower_left, Vec3 horizontal, Vec3 vertical,
        Vec3 origin, Hittables** world){
    int i = threadIdx.x + blockIdx.x  * blockDim.x;
    int j = threadIdx.y + blockIdx.y  * blockDim.y;

    if ((i >= maximum_x) || (j >= maximum_y)){
        return;
    }
    int pixel_index = j * maximum_x * 3 + i;
    float u = float(i) / float(maximum_x);
    float v = float(j) / float(maximum_y);
    Ray r(origin, lower_left + u * horizontal + v * vertical);
    fb[pixel_index] = ray_color(r, world);
}

__global__ void make_world(Hittables** world, Hittable** ss, int size){
    if (threadIdx.x == 0 && blockIdx.x == 0){
        // declare objects
        ss[0] = new Sphere(Vec3(0,0,-1), 0.5);
        ss[1] = new Sphere(Vec3(0,-100.5,-1), 100);
        // thrust::device_ptr<Hittable*> hs = thrust::device_malloc<Hittable*>(2);
        world[0] = new Hittables( ss, size);

    }
}
__global__ void free_world(Hittables** world,Hittable **ss){
    delete ss[0];
    delete ss[1];
    delete world;
}

int main(){
    int WIDTH = 1200;
    int HEIGHT = 600;
    int BLOCK_WIDTH = 8;
    int BLOCK_HEIGHT = 8;

    std::cerr << "Resim boyutumuz " << WIDTH << "x"
        << HEIGHT << std::endl;

    std::cerr << BLOCK_WIDTH << "x" << BLOCK_HEIGHT << " bloklar halinde"
        << std::endl;


    // declare frame size
    int total_pixel_size = WIDTH * HEIGHT;
    size_t frameSize = 3 * total_pixel_size;

    // declare frame
    thrust::device_ptr<Vec3> fb = thrust::device_new<Vec3>(frameSize);

    // declare world
    thrust::device_ptr<Hittables*> world = thrust::device_malloc<Hittables*>(1);
    thrust::device_ptr<Hittable*> hs = thrust::device_malloc<Hittable*>(2);



    make_world<<<1,1>>>(
            thrust::raw_pointer_cast(world),
            thrust::raw_pointer_cast(hs),
            2
            );
    CUDA_CONTROL(hipGetLastError());
    CUDA_CONTROL(hipDeviceSynchronize());

    clock_t baslar, biter;
    baslar = clock();

    dim3 blocks(WIDTH / BLOCK_WIDTH + 1,
            HEIGHT / BLOCK_HEIGHT + 1);
    dim3 threads(BLOCK_WIDTH, BLOCK_HEIGHT); 
    render<<<blocks, threads>>>(
            thrust::raw_pointer_cast(fb), 
            WIDTH, 
            HEIGHT,
            Vec3(-2.0f, -1.0f, -1.0f),
            Vec3(4.0f, 0.0f, 0.0f),
            Vec3(0.0f, 2.0f, 0.0f),
            Vec3(0.0f),
            thrust::raw_pointer_cast(world)
            );
    CUDA_CONTROL(hipGetLastError());
    CUDA_CONTROL(hipDeviceSynchronize());
    biter = clock();
    double saniyeler = ((double)(biter - baslar)) / CLOCKS_PER_SEC;
    std::cerr << "Islem " << saniyeler << " saniye surdu" 
        << std::endl;

    std::cout << "P3" << std::endl;
    std::cout << WIDTH << " " << HEIGHT << std::endl;
    std::cout << "255" << std::endl;

    for (int j = HEIGHT - 1; j >= 0; j--){
        for (int i = 0; i < WIDTH; i++){
            size_t pixel_index = j*3*WIDTH + i;
            thrust::device_reference<Vec3> pix_ref = fb[pixel_index];
            Vec3 pixel = pix_ref;
            int ir = int(255.99 * pixel.r());
            int ig = int(255.99 * pixel.g());
            int ib = int(255.99 * pixel.b());
            std::cout << ir << " " << ig << " "
                << ib << std::endl;
        }
    }
    CUDA_CONTROL(hipDeviceSynchronize());
    free_world<<<1,1>>>(
            thrust::raw_pointer_cast(world),
            thrust::raw_pointer_cast(hs)
            );
    CUDA_CONTROL(hipGetLastError());
    thrust::device_free(fb);
    CUDA_CONTROL(hipGetLastError());
    hipDeviceReset();
}
