#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include <oneweekend/vec3.hpp>
#include <oneweekend/color.hpp>

void cuda_control(hipError_t res, const char *const fn,
        const char * const f, const int l){
    if (res != hipSuccess){
        std::cerr << "CUDA ERROR :: " 
            << static_cast<unsigned int>(res)
            << " "
            << hipGetErrorName(res)
            << " file: " << f << " line: " << l 
            << " function: " << fn << std::endl;
        hipDeviceReset();
        exit(99);
    }
}

#define CUDA_CONTROL(v) cuda_control((v), #v, __FILE__, __LINE__)

__global__ void render(Vec3 *fb, int maximum_x, int maximum_y){
    int i = threadIdx.x + blockIdx.x  * blockDim.x;
    int j = threadIdx.y + blockIdx.y  * blockDim.y;

    if ((i >= maximum_x) || (j >= maximum_y)){
        return;
    }
    int pixel_index = j * maximum_x * 3 + i;
    fb[pixel_index] = Vec3(float(i) / maximum_x, float(j)/maximum_y, 0.1f);
}

int main(){
    int WIDTH = 1200;
    int HEIGHT = 600;
    int BLOCK_WIDTH = 8;
    int BLOCK_HEIGHT = 8;

    std::cerr << "Resim boyutumuz " << WIDTH << "x"
        << HEIGHT << std::endl;

    std::cerr << BLOCK_WIDTH << "x" << BLOCK_HEIGHT << " bloklar halinde"
        << std::endl;

    
    // declare frame
    Vec3 *fb;

    // declare frame size
    int total_pixel_size = WIDTH * HEIGHT;
    size_t frameSize = 3 * total_pixel_size * sizeof(Vec3);

    CUDA_CONTROL(
            hipMallocManaged(
                (void **)&fb, frameSize
                )
            );

    clock_t baslar, biter;
    baslar = clock();

    dim3 blocks(WIDTH / BLOCK_WIDTH + 1,
            HEIGHT / BLOCK_HEIGHT + 1);
    dim3 threads(BLOCK_WIDTH, BLOCK_HEIGHT); 
    render<<<blocks, threads>>>(fb, WIDTH, HEIGHT);
    CUDA_CONTROL(hipGetLastError());
    CUDA_CONTROL(hipDeviceSynchronize());
    biter = clock();
    double saniyeler = ((double)(biter - baslar)) / CLOCKS_PER_SEC;
    std::cerr << "Islem " << saniyeler << " saniye surdu" 
        << std::endl;

    std::cout << "P3" << std::endl;
    std::cout << WIDTH << " " << HEIGHT << std::endl;
    std::cout << "255" << std::endl;

    for (int j = HEIGHT - 1; j >= 0; j--){
        for (int i = 0; i < WIDTH; i++){
            size_t pixel_index = j*3*WIDTH + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " "
                << ib << std::endl;
        }
    }
    CUDA_CONTROL(hipFree(fb));
}
