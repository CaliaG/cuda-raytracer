#include "hip/hip_runtime.h"
#include <oneweekend/camera.cuh>
#include <oneweekend/color.hpp>
#include <oneweekend/debug.hpp>
#include <oneweekend/external.hpp>
#include <oneweekend/hittables.cuh>
#include <oneweekend/material.cuh>
#include <oneweekend/ray.cuh>
#include <oneweekend/sphere.cuh>
#include <oneweekend/vec3.cuh>

__device__ Color ray_color(const Ray &r, Hittables **world,
                           hiprandState *local_rand_state,
                           int bounceNb) {
  Ray current_ray = r;
  Vec3 current_attenuation = Vec3(1.0f);
  while (bounceNb > 0) {
    HitRecord rec;
    bool anyHit =
        world[0]->hit(current_ray, 0.001f, FLT_MAX, rec);
    if (anyHit) {
      Ray scattered;
      Vec3 attenuation;
      bool isScattered = rec.mat_ptr->scatter(
          current_ray, rec, attenuation, scattered,
          local_rand_state);
      if (isScattered) {
        bounceNb--;
        current_attenuation *= attenuation;
        current_ray = scattered;
      } else {
        return Vec3(0.0f); // background color
      }
    } else {
      Vec3 udir = to_unit(current_ray.direction());
      double t = 0.5f * (udir.y() + 1.0f);
      Vec3 out = (1.0f - t) * Vec3(1.0f) +
                 t * Vec3(0.5f, 0.7f, 1.0f);
      return current_attenuation * out;
    }
  }
  return Vec3(0.0f); // background color
}

__global__ void rand_init(hiprandState *randState,
                          int seed) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprand_init(seed, 0, 0, randState);
  }
}

__global__ void render_init(int mx, int my,
                            hiprandState *randState,
                            int seed) {
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    hiprand_init(seed, 0, 0, randState);
  }
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if ((i >= mx) || (j >= my)) {
    return;
  }
  int pixel_index = j * mx + i;
  // same seed, different index
  hiprand_init(seed, pixel_index, 0,
              &randState[pixel_index]);
}

__global__ void render(Vec3 *fb, int maximum_x,
                       int maximum_y, int sample_nb,
                       int bounceNb, Camera **cam,
                       Hittables **world,
                       hiprandState *randState) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if ((i >= maximum_x) || (j >= maximum_y)) {
    return;
  }
  int pixel_index = j * maximum_x + i;
  hiprandState localS = randState[pixel_index];
  Vec3 rcolor(0.0f);
  for (int s = 0; s < sample_nb; s++) {
    double u = double(i + hiprand_uniform(&localS)) /
              double(maximum_x);
    double v = double(j + hiprand_uniform(&localS)) /
              double(maximum_y);
    Ray r = cam[0]->get_ray(u, v, &localS);
    rcolor += ray_color(r, world, randState, bounceNb);
  }
  // fix the bounce depth
  randState[pixel_index] = localS;
  rcolor /= double(sample_nb);
  rcolor.e[0] = sqrt(rcolor.x());
  rcolor.e[1] = sqrt(rcolor.y());
  rcolor.e[2] = sqrt(rcolor.z());
  fb[pixel_index] = rcolor;
}

__global__ void make_world(Hittables **world, Hittable **ss,
                           int size, Camera **cam, int nx,
                           int ny, hiprandState *randState,
                           int row) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    // declare objects
    Lambertian *lamb = new Lambertian(Vec3(0.5, 0.5, 0.5));
    ss[0] = new Sphere(Vec3(0, -1000.0, -1), 1000, lamb);
    int i = 1;
    int halfRow = row / 2;
    for (int a = -halfRow; a < halfRow; a++) {
      for (int b = -halfRow; b < halfRow; b++) {
        double choose_mat = hiprand_uniform(randState);
        Vec3 center(a + hiprand_uniform(randState), 0.2,
                    b + hiprand_uniform(randState));
        if (choose_mat < 0.8f) {
          Point3 center2 =
              center +
              Vec3(0, random_double(randState, 0.0, 0.5), 0);
          Color albedo = random_double(randState);
          albedo *= random_double(randState);
          Material *lamb1 = new Lambertian(albedo);
          ss[i++] = new MovingSphere(center, center2, 0.0,
                                     1.0, 0.2, lamb1);
        } else if (choose_mat < 0.95f) {
          Material *met = new Metal(
              Vec3(
                  0.5f * (1.0f + hiprand_uniform(randState)),
                  0.5f * (1.0f + hiprand_uniform(randState)),
                  0.5f *
                      (1.0f + hiprand_uniform(randState))),
              0.5f * hiprand_uniform(randState));
          ss[i++] = new Sphere(center, 0.2, met);
        } else {
          Material *diel = new Dielectric(1.5);
          ss[i++] = new Sphere(center, 0.2, diel);
        }
      }
    }

    Material *diel = new Dielectric(1.5);
    ss[i++] = new Sphere(Vec3(0, 1, 0), 1.0, diel);

    Material *lamb2 = new Lambertian(Vec3(0.4, 0.2, 0.1));
    ss[i++] = new Sphere(Vec3(-4, 1, 0), 1.0, lamb2);

    Material *met2 = new Metal(Vec3(0.7, 0.6, 0.5), 0.0);
    ss[i++] = new Sphere(Vec3(4, 1, 0), 1.0, met2);
    world[0] = new Hittables(ss, 22 * 22 + 1 + 3);

    Vec3 lookfrom(13, 2, 3);
    Vec3 lookat(0, 0, 0);
    double dist_to_focus = 10.0;
    (lookfrom - lookat).length();
    double aperture = 0.1;
    cam[0] = new Camera(lookfrom, lookat, Vec3(0, 1, 0),
                        20.0, double(nx) / double(ny),
                        aperture, dist_to_focus, 0.0, 1.0);
  }
}
__global__ void free_world(Hittables **world, Hittable **ss,
                           Camera **cam) {
  int size = 22 * 22 + 1 + 3;
  for (int i = 0; i < size; i++) {
    delete ((Hittable *)ss[i])->mat_ptr;
    delete ss[i];
  }
  delete world[0];
  delete cam[0];
}

int main() {
  double aspect_ratio = 16.0f / 9.0f;
  int WIDTH = 320;
  int HEIGHT = static_cast<int>(WIDTH / aspect_ratio);
  int BLOCK_WIDTH = 10;
  int BLOCK_HEIGHT = 10;
  int SAMPLE_NB = 30;
  int BOUNCE_NB = 20;

  std::cerr << "Resim boyutumuz " << WIDTH << "x" << HEIGHT
            << std::endl;

  std::cerr << BLOCK_WIDTH << "x" << BLOCK_HEIGHT
            << " bloklar halinde" << std::endl;

  // declare frame size
  int total_pixel_size = WIDTH * HEIGHT;
  size_t frameSize = 3 * total_pixel_size;

  // declare frame
  thrust::device_ptr<Vec3> fb =
      thrust::device_malloc<Vec3>(frameSize);
  CUDA_CONTROL(hipGetLastError());

  // declare random state
  int SEED = 1987;
  thrust::device_ptr<hiprandState> randState1 =
      thrust::device_malloc<hiprandState>(frameSize);
  CUDA_CONTROL(hipGetLastError());

  // declare random state 2
  thrust::device_ptr<hiprandState> randState2 =
      thrust::device_malloc<hiprandState>(1);
  CUDA_CONTROL(hipGetLastError());
  rand_init<<<1, 1>>>(thrust::raw_pointer_cast(randState2),
                      SEED);
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  // declare world
  thrust::device_ptr<Hittables *> world =
      thrust::device_malloc<Hittables *>(1);
  CUDA_CONTROL(hipGetLastError());
  int row = 22;
  int focus_obj_nb = 3;
  int nb_hittable = row * row + 1 + focus_obj_nb;
  thrust::device_ptr<Hittable *> hs =
      thrust::device_malloc<Hittable *>(nb_hittable);
  CUDA_CONTROL(hipGetLastError());

  // declare camera
  thrust::device_ptr<Camera *> cam =
      thrust::device_malloc<Camera *>(1);
  CUDA_CONTROL(hipGetLastError());

  make_world<<<1, 1>>>(
      thrust::raw_pointer_cast(world),
      thrust::raw_pointer_cast(hs), 2,
      thrust::raw_pointer_cast(cam), WIDTH, HEIGHT,
      thrust::raw_pointer_cast(randState2), row);
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  clock_t baslar, biter;
  baslar = clock();

  dim3 blocks(WIDTH / BLOCK_WIDTH + 1,
              HEIGHT / BLOCK_HEIGHT + 1);
  dim3 threads(BLOCK_WIDTH, BLOCK_HEIGHT);
  render_init<<<blocks, threads>>>(
      WIDTH, HEIGHT, thrust::raw_pointer_cast(randState1),
      SEED + 7);
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());

  render<<<blocks, threads>>>(
      thrust::raw_pointer_cast(fb), WIDTH, HEIGHT,
      SAMPLE_NB, BOUNCE_NB, thrust::raw_pointer_cast(cam),
      thrust::raw_pointer_cast(world),
      thrust::raw_pointer_cast(randState1));
  CUDA_CONTROL(hipGetLastError());
  CUDA_CONTROL(hipDeviceSynchronize());
  biter = clock();
  double saniyeler =
      ((double)(biter - baslar)) / CLOCKS_PER_SEC;
  std::cerr << "Islem " << saniyeler << " saniye surdu"
            << std::endl;

  std::cout << "P3" << std::endl;
  std::cout << WIDTH << " " << HEIGHT << std::endl;
  std::cout << "255" << std::endl;

  for (int j = HEIGHT - 1; j >= 0; j--) {
    for (int i = 0; i < WIDTH; i++) {
      size_t pixel_index = j * WIDTH + i;
      thrust::device_reference<Vec3> pix_ref =
          fb[pixel_index];
      Vec3 pixel = pix_ref;
      int ir = int(255.99 * pixel.r());
      int ig = int(255.99 * pixel.g());
      int ib = int(255.99 * pixel.b());
      std::cout << ir << " " << ig << " " << ib
                << std::endl;
    }
  }
  CUDA_CONTROL(hipDeviceSynchronize());
  free_world<<<1, 1>>>(thrust::raw_pointer_cast(world),
                       thrust::raw_pointer_cast(hs),
                       thrust::raw_pointer_cast(cam));
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(fb);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(world);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(hs);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(cam);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(randState2);
  CUDA_CONTROL(hipGetLastError());
  thrust::device_free(randState1);
  CUDA_CONTROL(hipGetLastError());
  hipDeviceReset();
}
