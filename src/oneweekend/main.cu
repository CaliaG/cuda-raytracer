#include "hip/hip_runtime.h"
#include <oneweekend/vec3.cuh>
#include <oneweekend/color.hpp>
#include <oneweekend/ray.cuh>
#include <oneweekend/sphere.cuh>
#include <oneweekend/hittables.cuh>
#include <oneweekend/camera.cuh>
#include <oneweekend/debug.hpp>
#include <oneweekend/external.hpp>


__device__ Color ray_color(
        const Ray & r,
        Hittables** world, 
        hiprandState *local_rand_state,
        int bounceNb){
    Ray current_ray = r;
    float current_attenuation = 1.0f;
    while (bounceNb > 0){
        HitRecord rec;
        bool anyHit = world[0]->hit(current_ray, 0.001f, FLT_MAX, rec);
        if (anyHit){
            Ray scattered;
            Vec3 attenuation;
            bool isScattered = rec.mat_ptr->scatter(current_ray, 
                    attenuation, scattered, local_rand_state);
            if (isScattered){
                bounceNb--;
                current_attenuation *= attenuation;
                current_ray = scattered;
            }else{
                return Vec3(0.0f); // background color
            }
        } else {
            Vec3 udir = to_unit(current_ray.direction());
            float t = 0.5f * (udir.y() + 1.0f);
            Vec3 out = (1.0f-t)*Vec3(1.0f)+ t*Vec3(0.5f, 0.7f, 1.0f);
            return current_attenuation * out;
        }
    }
    return Vec3(0.0f); // background color

}

__global__ void render_init(hiprandState *randState){
    if (threadIdx.x == 0 && threadIdx.y == 0){
        curand_int(1923, 0,0, randState);
    }
    int i = threadIdx.x + blockIdx.x  * blockDim.x;
    int j = threadIdx.y + blockIdx.y  * blockDim.y;

    if ((i >= mx) || (j >= my)){
        return;
    }
    int pixel_index = j * mx * 3 + i;
    // same seed, different index
    hiprand_init(1987, pixel_index, 0, &randState[pixel_index]);
}

__global__ void render(
        Vec3 *fb, int maximum_x, int maximum_y, int sample_nb, int bounceNb,
        Camera** cam,
        Hittables** world,
        hiprandState *randState){
    int i = threadIdx.x + blockIdx.x  * blockDim.x;
    int j = threadIdx.y + blockIdx.y  * blockDim.y;

    if ((i >= maximum_x) || (j >= maximum_y)){
        return;
    }
    int pixel_index = j * maximum_x * 3 + i;
    hiprandState localS = randState[pixel_index];
    Vec3 rcolor(0.0f);
    for(int s = 0; s < sample_nb; s++){
        float u = float(i + hiprand_uniform(&localS)) / float(maximum_x);
        float v = float(j+ hiprand_uniform(&localS)) / float(maximum_y);
        Ray r = cam[0]->get_ray(u,v);
        rcolor += ray_color(r, world, randState, bounceNb);
    }
    // fix the bounce depth
    randState[pixel_index] = localS;
    rcolor /= float(sample_nb);
    rcolor.e[0] = sqrt(rcolor.x());
    rcolor.e[1] = sqrt(rcolor.y());
    rcolor.e[2] = sqrt(rcolor.z());
    fb[pixel_index] = rcolor;
}


__global__ void make_world(Hittables** world, Hittable** ss, int size,
        Camera** cam, int nx, int ny, hiprandState * randState){
    if (threadIdx.x == 0 && blockIdx.x == 0){
        // declare objects
        hiprandState local_rand_state = *rand_state;
        ss[0] = new Sphere(vec3(0,-1000.0,-1), 1000,
                new Lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                Vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    ss[i++] = new Sphere(center, 0.2,
                            new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    ss[i++] = new Sphere(center, 0.2,
                            new Metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    ss[i++] = new Sphere(center, 0.2, new Dielectric(1.5));
                }
            }
        }
        ss[i++] = new Sphere(Vec3(0, 1,0),  1.0, new Dielectric(1.5));
        ss[i++] = new Sphere(Vec3(-4, 1, 0), 1.0, new Lambertian(vec3(0.4, 0.2, 0.1)));
        ss[i++] = new Sphere(Vec3(4, 1, 0),  1.0, new Metal(vec3(0.7, 0.6, 0.5), 0.0));
        rand_state[0] = local_rand_state;
        world[0]  = new Hittables(d_list, 22*22+1+3);

        Vec3 lookfrom(13,2,3);
        Vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        cam[0]   = new camera(lookfrom,
                lookat,
                Vec3(0,1,0),
                30.0,
                float(nx)/float(ny),
                aperture,
                dist_to_focus);
    }
}
__global__ void free_world(Hittables** world,Hittable **ss, Camera**cam, int size=22*22+1+3){
    for(int i=0; i < size; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete ss[i];
    }
    delete ss[0];
    delete ss[1];
    delete world[0];
    delete cam[0];
}

int main(){
    int WIDTH = 1200;
    int HEIGHT = 600;
    int BLOCK_WIDTH = 8;
    int BLOCK_HEIGHT = 8;
    int SAMPLE_NB = 120;
    int BOUNCE_NB = 50;

    std::cerr << "Resim boyutumuz " << WIDTH << "x"
        << HEIGHT << std::endl;

    std::cerr << BLOCK_WIDTH << "x" << BLOCK_HEIGHT << " bloklar halinde"
        << std::endl;


    // declare frame size
    int total_pixel_size = WIDTH * HEIGHT;
    size_t frameSize = 3 * total_pixel_size;

    // declare frame
    thrust::device_ptr<Vec3> fb = thrust::device_malloc<Vec3>(frameSize);
    CUDA_CONTROL(hipGetLastError());

    // declare random state
    thrust::device_ptr<hiprandState> randState = thrust::device_malloc<hiprandState>(frameSize);
    CUDA_CONTROL(hipGetLastError());

    // declare world
    thrust::device_ptr<Hittables*> world = thrust::device_malloc<Hittables*>(1);
    CUDA_CONTROL(hipGetLastError());
    thrust::device_ptr<Hittable*> hs = thrust::device_malloc<Hittable*>(2);
    CUDA_CONTROL(hipGetLastError());

    // declare camera
    thrust::device_ptr<Camera*> cam = thrust::device_malloc<Camera*>(1);
    CUDA_CONTROL(hipGetLastError());

    make_world<<<1,1>>>(
            thrust::raw_pointer_cast(world),
            thrust::raw_pointer_cast(hs),
            2,
            thrust::raw_pointer_cast(cam)
            );
    CUDA_CONTROL(hipGetLastError());
    CUDA_CONTROL(hipDeviceSynchronize());

    clock_t baslar, biter;
    baslar = clock();

    dim3 blocks(WIDTH / BLOCK_WIDTH + 1,
            HEIGHT / BLOCK_HEIGHT + 1);
    dim3 threads(BLOCK_WIDTH, BLOCK_HEIGHT); 
    render_init<<<blocks, threads>>>(
            WIDTH, 
            HEIGHT,
            thrust::raw_pointer_cast(randState)
            );
    CUDA_CONTROL(hipGetLastError());
    CUDA_CONTROL(hipDeviceSynchronize());

    render<<<blocks, threads>>>(
            thrust::raw_pointer_cast(fb), 
            WIDTH, 
            HEIGHT,
            SAMPLE_NB,
            BOUNCE_NB,
            thrust::raw_pointer_cast(cam),
            thrust::raw_pointer_cast(world),
            thrust::raw_pointer_cast(randState)
            );
    CUDA_CONTROL(hipGetLastError());
    CUDA_CONTROL(hipDeviceSynchronize());
    biter = clock();
    double saniyeler = ((double)(biter - baslar)) / CLOCKS_PER_SEC;
    std::cerr << "Islem " << saniyeler << " saniye surdu" 
        << std::endl;

    std::cout << "P3" << std::endl;
    std::cout << WIDTH << " " << HEIGHT << std::endl;
    std::cout << "255" << std::endl;

    for (int j = HEIGHT - 1; j >= 0; j--){
        for (int i = 0; i < WIDTH; i++){
            size_t pixel_index = j*3*WIDTH + i;
            thrust::device_reference<Vec3> pix_ref = fb[pixel_index];
            Vec3 pixel = pix_ref;
            int ir = int(255.99 * pixel.r());
            int ig = int(255.99 * pixel.g());
            int ib = int(255.99 * pixel.b());
            std::cout << ir << " " << ig << " "
                << ib << std::endl;
        }
    }
    CUDA_CONTROL(hipDeviceSynchronize());
    free_world<<<1,1>>>(
            thrust::raw_pointer_cast(world),
            thrust::raw_pointer_cast(hs),
            thrust::raw_pointer_cast(cam)
            );
    CUDA_CONTROL(hipGetLastError());
    thrust::device_free(fb);
    CUDA_CONTROL(hipGetLastError());
    thrust::device_free(world);
    CUDA_CONTROL(hipGetLastError());
    thrust::device_free(hs);
    CUDA_CONTROL(hipGetLastError());
    thrust::device_free(cam);
    CUDA_CONTROL(hipGetLastError());
    thrust::device_free(randState);
    CUDA_CONTROL(hipGetLastError());
    hipDeviceReset();
}
